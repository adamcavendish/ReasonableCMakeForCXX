#include <cstdio>
#include <cstdlib>

#include <project6/cuda_util.hpp>

void cuda_eexit(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "cuda error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
