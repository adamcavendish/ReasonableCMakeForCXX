#include "hip/hip_runtime.h"
// STL
#include <algorithm>
// C-Standard
#include <cassert>
#include <cmath>
#include <cstdio>
#include <cstdlib>

#include <project6/cuda_util.hpp>

__global__ void vecAdd(const float *a, const float *b, float *c, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < n)
    c[id] = a[id] + b[id];
}

int main(int argc, char *argv[]) {
  int n = 100000;

  auto *h_a = new float[n];
  auto *h_b = new float[n];
  auto *h_c = new float[n];

  float *d_a = nullptr;
  float *d_b = nullptr;
  float *d_c = nullptr;
  cuda_eexit(hipMalloc(&d_a, sizeof(float)*n));
  cuda_eexit(hipMalloc(&d_b, sizeof(float)*n));
  cuda_eexit(hipMalloc(&d_c, sizeof(float)*n));

  for (int i = 0; i < n; i++) {
    h_a[i] = sin(i)*sin(i);
    h_b[i] = cos(i)*cos(i);
  }

  cuda_eexit(hipMemcpy(d_a, h_a, sizeof(float)*n, hipMemcpyHostToDevice));
  cuda_eexit(hipMemcpy(d_b, h_b, sizeof(float)*n, hipMemcpyHostToDevice));

  int threadsPerBlock = 256;
  int blocksPerGrid =(n + threadsPerBlock - 1) / threadsPerBlock;
  vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
  cuda_eexit(hipGetLastError());

  cuda_eexit(hipMemcpy(h_c, d_c, sizeof(float)*n, hipMemcpyDeviceToHost));

  float sum = std::accumulate(h_c, h_c + n, 0.f);
  printf("final result: %f\n", sum / n);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  delete h_a;
  delete h_b;
  delete h_c;

  return 0;
}
